#include "hip/hip_runtime.h"
//******************************************************************************
// Final Project
// Name: Andrew McKissick
// GPU Computing Date: 12/5/16
//******************************************************************************
// This program performs a Fast Fourier transform on a set of complex numbers.
// Input is read from a file (data.txt) as floating point pairs. Error checking
// is not performed on input data, so anything other than an even number less
// than or equal to 2*N of decimal numbers separated by whitespace will cause
// undefined behavior. Output is to stdout and limited to the first 8 terms,
// and the time required for program and funcion execution, though this is
// easily configurable in the code. No user interaction is required beyond
// creating a valid input file and running the program.
//******************************************************************************

#define _CRT_SECURE_NO_WARNINGS
#include <stdio.h>
#define _USE_MATH_DEFINES
#include <math.h>
#include <time.h>

//number of elements in each array
#define N 16384

__global__ void FFTCalc(double realIn[], double imagIn[], double realOut[], double imagOut[]);

int main()
{
	//clock_t prog_start, prog_end, calc_start, calc_end, read_start, read_end;
	float comm_time_1, comm_time_2, calc_time;
	hipEvent_t start, stop;
	
	//prog_start = clock();
	FILE *data;
	data = fopen("data.txt", "r");
	
	const int DATASIZE = N*sizeof(double);

	double xReal[N];
	double xImag[N];
	
	double* dev_xReal;
	double* dev_xImag;
	
	double* dev_XReal;
	double* dev_XImag;

	//read from data.txt into the real and imaginary arrays until EOF
	//read_start = clock();
	int i = 0;
	while (fscanf(data, "%lf", &xReal[i]) != EOF)
	{
		fscanf(data, "%lf", &xImag[i]);
		i++;
	}
	//read_end = clock();

	fclose(data);

	//fill the remaining indices in the real and imaginary arrays with 0.0
	for (; i < N; i++)
	{
		xReal[i] = 0.0;
		xImag[i] = 0.0;
	}
	
	hipEventCreate(&start);
  hipEventRecord(start,0);
	
	hipMalloc((void**)&dev_xReal, DATASIZE);
	hipMalloc((void**)&dev_xImag, DATASIZE);
	
	hipMalloc((void**)&dev_XReal, DATASIZE);
	hipMalloc((void**)&dev_XImag, DATASIZE);
	
	hipMemcpy(dev_xReal, xReal, DATASIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_xImag, xImag, DATASIZE, hipMemcpyHostToDevice);
	
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&comm_time_1, start, stop);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	dim3 dimGrid(N/1024, 1); 	
	dim3 dimBlock(1024, 1);

	double XReal[N];
	double XImag[N];

	//calculate the FFT
	//calc_start = clock();
	
	hipEventCreate(&start);
  hipEventRecord(start,0);
	
	FFTCalc<<<dimGrid, dimBlock>>>(dev_xReal, dev_xImag, dev_XReal, dev_XImag);
	hipDeviceSynchronize();
	
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&calc_time, start, stop);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	//calc_end = clock();
	
	hipEventCreate(&start);
  hipEventRecord(start,0);
	
	hipMemcpy(XReal, dev_XReal, DATASIZE, hipMemcpyDeviceToHost);
	hipMemcpy(XImag, dev_XImag, DATASIZE, hipMemcpyDeviceToHost);
	
	hipFree(dev_xReal);
	hipFree(dev_xImag);
	
	hipFree(dev_XReal);
	hipFree(dev_XImag);
	
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&comm_time_2, start, stop);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	//prog_end = clock();
	
	//prog_time = (double)(prog_end - prog_start) / CLOCKS_PER_SEC;
	//calc_time = (double)(calc_end - calc_start) / CLOCKS_PER_SEC;
	//read_time = (double)(read_end - read_start) / CLOCKS_PER_SEC;

	//display the first 8 values
	printf("TOTAL PROCESSED SAMPLES: %d\n", N);
	printf("================================================================================\n");
	for (i = 0; i < 8; i++)
	{
		printf("XR[%d]: %f\n", i, XReal[i]);
		printf("XI[%d]: %f\n\n", i, XImag[i]);
	}
	printf("================================================================================\n\n");
	printf("Time required for calculation: %f\n", calc_time);
	printf("Time required for communication: %f\n", (comm_time_1 + comm_time_2));

	//printf("Time required for file read: %f\n", read_time);
	//printf("Time required for entire program: %f\n", prog_time);

	return 0;
}

//******************************************************************************
// FFTCalc()
// This kernel calculates the coefficients of the fourier transform. It requires
// two double arrays of size N containing the real and imaginary parts of the
// function to be transformed, and two double arrays for storing the real and
// imaginary parts of the result. No error check is performed so if these
// requirements are not met it is likely the program will crash. This program
// computes large sums in sequence to avoid data corrumption. While it is
// possible to make these calculations more parallel, I felt that given the
// number of data accesses this would require and the number of threads
// available on a node, that this would likely be counterproductive to
// efficiently calculating the fourier transform.
//******************************************************************************
__global__ void FFTCalc(double realIn[], double imagIn[], double realOut[], double imagOut[])
{
	//accumulators. e = even index, o = odd index.
	double eReal = 0, eImag = 0, oCosReal = 0, oCosImag = 0, oSinReal = 0, oSinImag = 0;

	//frequently used elements. Precalculated here or at the beginning of the loop
	double theta = (-2.0 * M_PI * (double)(threadIdx.x + blockIdx.x * 1024)) / (double)N;
	double cos2MTheta = 0, sin2MTheta = 0;
	int m = 0, twoM = 0, twoMPlus = 0;

	for (; m < (N / 2); m++)
	{
		twoM = 2 * m;
		twoMPlus = twoM + 1;
		cos2MTheta = cos((double)twoM * theta);
		sin2MTheta = sin((double)twoM * theta);

		eReal += ((realIn[twoM] * cos2MTheta) - (imagIn[twoM] * sin2MTheta));
		eImag += ((realIn[twoM] * sin2MTheta) + (imagIn[twoM] * cos2MTheta));
		oCosReal += ((realIn[twoMPlus] * cos2MTheta) - (imagIn[twoMPlus] * sin2MTheta));
		oCosImag += ((realIn[twoMPlus] * sin2MTheta) + (imagIn[twoMPlus] * cos2MTheta));
		oSinReal -= ((realIn[twoMPlus] * sin2MTheta) + (imagIn[twoMPlus] * cos2MTheta));
		oSinImag += ((realIn[twoMPlus] * cos2MTheta) - (imagIn[twoMPlus] * sin2MTheta));
	}
	realOut[threadIdx.x + blockIdx.x * 1024] = eReal + cos(theta) * oCosReal + sin(theta) * oSinReal;
	imagOut[threadIdx.x + blockIdx.x * 1024] = eImag + cos(theta) * oCosImag + sin(theta) * oSinImag;

	return;
}